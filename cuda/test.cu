#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <chrono>
#include "../matrix_io.h"
#include "../util.h"
#include "timer.h"

using namespace std;
using Clock=std::chrono::high_resolution_clock;

#define HASH_TABLE_SIZE 32


// Define key(i, j), convert coordinate(i, j) to a size_t value
__device__ inline int key(int i, int j, int COLS) { return i * COLS + j;}
inline int get_i(int index, int COLS) { return index / COLS; }
inline int get_j(int index, int COLS) { return index % COLS; }


// Every thread in the same block will run this method with different scalar
// So mat better be stored in shared memory
__device__
inline void scale_csr_row(Matrix mat, float scalar, int A_row, int A_col, int *dev_C_key, float *dev_C_val) {
    // access ith row of B, i.e. ith col of A
    for(int i=mat.I[A_col]; i < mat.I[A_col+1]; i++) {
        int B_col = mat.J[i];
        float val = mat.val[i] * scalar;

        // p is a simple hash code for (A_row, B_col) pair
        int p = key(A_row, B_col, mat.M);
        int index = p & (HASH_TABLE_SIZE-1);

        dev_C_key[index] = p;
        atomicAdd(&(dev_C_val[index]), val);

        //printf("p:%d, index:%d, (%d %d %f %f)\n", p, index, A_row, B_col, val, dev_C_val[index]);
    }
}

__global__
void cuda_multiplication(Matrix A, Matrix B, int *dev_C_key, float *dev_C_val) {
    /*
     *  go through all columns of csc matrix A
     *  we set blockDim.x == A.N
     *  so each block is responsible for processing one column of A
     */

    //for (int i = 0; i < A.N; i++) {
    int i = blockIdx.x;
        // get all values in ith column of A
        // each thread in the same block get a portion of tasks
        int load = (A.J[i+1] - A.J[i]) / blockDim.x;
        if((A.J[i+1] - A.J[i]) % blockDim.x !=0) load = load + 1;
        for(int j = 0; j < load; j++) {
            int k = A.J[i] + threadIdx.x * load + j;
            if(k < A.J[i+1]) {
                int row = A.I[k];
                float scalar = A.val[k];
                scale_csr_row(B, scalar, row, i, dev_C_key, dev_C_val);
            }
        }
    //}
}


int main(int argc, char *argv[]) {
    if (argc < 3) {
        printf("Usage: %s [martix-market-filename] [matrix-market-filename]\n", argv[0]);
        exit(1);
    }

    // Read matrix from matrix-market file
    Matrix A, B;
    read_mm_matrix_csc(argv[1], &(A.M), &(A.N), &(A.nz), &(A.I), &(A.J), &(A.val));
    print_matrix_head(A);
    read_mm_matrix_csr(argv[2], &(B.M), &(B.N), &(B.nz), &(B.I), &(B.J), &(B.val));
    print_matrix_head(B);


    // Allocate device memory and copy matrices from the host to the device
    timer t1;
    Matrix dev_A, dev_B;
    dev_A.M = A.M; dev_A.N = A.N; dev_A.nz = A.nz;
    hipMalloc((void**)&dev_A.I, sizeof(int)*(A.nz));
    hipMalloc((void**)&dev_A.J, sizeof(int)*(A.N+1));
    hipMalloc((void**)&dev_A.val, sizeof(float)*(A.nz));
    hipMemcpy(dev_A.I, A.I, A.nz*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_A.J, A.J, (A.N+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_A.val, A.val, A.nz*sizeof(float), hipMemcpyHostToDevice);
    dev_B.M = B.M; dev_B.N = B.N; dev_B.nz = B.nz;
    hipMalloc((void**)&dev_B.I, sizeof(int)*(B.M+1));
    hipMalloc((void**)&dev_B.J, sizeof(int)*(B.nz));
    hipMalloc((void**)&dev_B.val, sizeof(float)*(B.nz));
    hipMemcpy(dev_B.I, B.I, (B.M+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_B.J, B.J, B.nz*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_B.val, B.val, B.nz*sizeof(float), hipMemcpyHostToDevice);

    // Allocate device memory for has table
    int *dev_C_key;
    float *dev_C_val;
    hipMalloc((void**)&dev_C_key, sizeof(int)*HASH_TABLE_SIZE);
    hipMemset(dev_C_key, -1, sizeof(int)*HASH_TABLE_SIZE);
    hipMalloc((void**)&dev_C_val, sizeof(float)*HASH_TABLE_SIZE);
    hipMemset(dev_C_val, 0, sizeof(float)*HASH_TABLE_SIZE);
    printf("time for allocating memory:%f milliseconds\n", t1.milliseconds_elapsed());

    timer t2;
    cuda_multiplication<<<A.N, 32>>>(dev_A, dev_B, dev_C_key, dev_C_val);
    printf("time for SpMM: %f milliseconds\n", t2.milliseconds_elapsed());

    // Copy back the hash table
    int *C_key = new int[HASH_TABLE_SIZE];
    float *C_val = new float[HASH_TABLE_SIZE];
    hipMemcpy(C_key, dev_C_key, HASH_TABLE_SIZE*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(C_val, dev_C_val, HASH_TABLE_SIZE*sizeof(float), hipMemcpyDeviceToHost);
    for(int i = 0; i < HASH_TABLE_SIZE; i++) {
        int index = C_key[i];
        if(index != -1)
            printf("%d: (%d %d %f)\n", index, get_i(index, A.N), get_j(index, A.N), C_val[i]);
    }



    hipFree(dev_A.I);
    hipFree(dev_A.J);
    hipFree(dev_A.val);
    hipFree(dev_B.I);
    hipFree(dev_B.J);
    hipFree(dev_B.val);
    hipFree(dev_C_key);
    hipFree(dev_C_val);
    delete C_key;
    delete C_val;
    return 0;
}
