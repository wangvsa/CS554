#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <chrono>
#include "../matrix_io.h"
#include "../util.h"
#include "timer.h"

using namespace std;
using Clock=std::chrono::high_resolution_clock;

// Define key(i, j), convert coordinate(i, j) to a size_t value
inline size_t key(int i, int j) {return (size_t) i << 32 | (unsigned int) j;}
inline int get_first(size_t C) { return C>>32; }
inline int get_second(size_t C) { return C & 0xFFFFFFFF; }


// Every thread in the same block will work on this method with different scalar
// So mat should be stored in shared memory
__device__
void scale_csr_row(Matrix mat, float scalar, int A_row, int A_col) {
    // access ith row of B, i.e. ith col of A
    for(int i=mat.I[A_col]; i < mat.I[A_col+1]; i++) {
        int B_col = mat.J[i];
        double val = mat.val[i] * scalar;
        //size_t p = key(A_row, B_col);
        /*
        if((*C).find(p) == (*C).end()) {
            (*C)[p] = val;
        } else {
            (*C)[p] += val;
        }
        */
    }
}

__global__
void cuda_multiplication(Matrix A, Matrix B) {
    // go through all columns of csc matrix A
    // each block is responsible for processing one column of A
    for (int i = 0; i < A.N; i++) {
        // get all values in ith column of A
        for(int j = A.J[i]; j < A.J[i+1]; j++) {
            int row = A.I[j];
            float scalar = A.val[j];
            scale_csr_row(B, scalar, row, i);
        }
    }
}


int main(int argc, char *argv[]) {
    Matrix A, B;
    if (argc < 3) {
        printf("Usage: %s [martix-market-filename]\n", argv[0]);
        exit(1);
    }

    read_mm_matrix_csc(argv[1], &(A.M), &(A.N), &(A.nz), &(A.I), &(A.J), &(A.val));
    print_matrix_head(A);
    read_mm_matrix_csr(argv[2], &(B.M), &(B.N), &(B.nz), &(B.I), &(B.J), &(B.val));
    print_matrix_head(B);

    timer t;
    cuda_multiplication<<<10, 128>>>(A, B);
    printf("time:%f milliseconds\n", t.milliseconds_elapsed());

    return 0;
}
