#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <chrono>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "../matrix_io.h"
#include "../util.h"
#include "esc.h"
#include "timer.h"

using namespace std;
using Clock=std::chrono::high_resolution_clock;


#define HASH_TABLE_SIZE 268435456
#define GRID_SIZE 10000                // the number of blocks
#define BLOCK_SIZE 32                  // the number of threads in each block
int const INVALID_KEY = 0;
int BLOCK_CONFLICT_SIZE = 100;        // the size of confict array for each block


// For storing collision results
typedef struct ConflictArray_t {
    int *counter;     // each block has its own conflict counter
    unsigned int *key;
    float *val;
} ConflictArray;


// Hash function to convert coordinate(i, j) to a single integer
__device__ inline unsigned int hashcode(int i, int j, int COLS) { return i * COLS + j; }
inline int get_i(unsigned int key, int COLS) { return (key-1) / COLS; }
inline int get_j(unsigned int key, int COLS) { return (key-1) % COLS; }


// Every thread in the same block will run this method with different scalar
// So mat better be stored in shared memory
__device__
inline void scale_csr_row(Matrix mat, float scalar, int A_row, int A_col, unsigned int *dev_C_key, float *dev_C_val, ConflictArray conflict_array, int BLOCK_CONFLICT_SIZE) {
    // access ith row of B, i.e. ith col of A
    for(int i=mat.I[A_col]; i < mat.I[A_col+1]; i++) {
        int B_col = mat.J[i];
        float val = mat.val[i] * scalar;

        // p is a simple hash code for (A_row, B_col) pair
        unsigned int h = hashcode(A_row, B_col, mat.M) + 1;      // leave zero as the INVALID_KEY
        unsigned int index = h & (HASH_TABLE_SIZE-1);

        unsigned int old_h = atomicCAS(&(dev_C_key[index]), INVALID_KEY, h);
        if(old_h == INVALID_KEY) {
            atomicAdd(&(dev_C_val[index]), val);
        } else {
            if(old_h == h) {
                atomicAdd(&(dev_C_val[index]), val);
            } else {
                int counter = atomicAdd(&(conflict_array.counter[blockIdx.x]), 1);
                atomicExch(&(conflict_array.key[blockIdx.x*BLOCK_CONFLICT_SIZE+counter]), h);
                atomicExch(&(conflict_array.val[blockIdx.x*BLOCK_CONFLICT_SIZE+counter]), val);
            }
        }
        //printf("h:%d, index:%d, (%d %d %f %f)\n", h, index, A_row, B_col, val, dev_C_val[index]);
    }
}

__global__
void cuda_multiplication(Matrix A, Matrix B, unsigned int *dev_C_key, float *dev_C_val, ConflictArray conflict_array, int BLOCK_CONFLICT_SIZE) {
    /*
     *  go through all columns of csc matrix A
     *  each block is responsible for processing one column of A
     */
    int block_load = A.N / gridDim.x;
    if(A.N % gridDim.x != 0) block_load = block_load + 1;

    for (int i = blockIdx.x*block_load; i < (blockIdx.x+1)*block_load; i++) {
        if(i >= A.N) return;
        // get all values in ith column of A
        // each thread in the same block get a portion of tasks
        int load = (A.J[i+1] - A.J[i]) / blockDim.x;
        if((A.J[i+1] - A.J[i]) % blockDim.x !=0) load = load + 1;
        for(int j = 0; j < load; j++) {
            int k = A.J[i] + threadIdx.x * load + j;
            if(k < A.J[i+1]) {
                int row = A.I[k];
                float scalar = A.val[k];
                scale_csr_row(B, scalar, row, i, dev_C_key, dev_C_val, conflict_array, BLOCK_CONFLICT_SIZE);
            }
        }
    }

    // Debug, print conflict for each block
    /*
    if(blockIdx.x == 0 && threadIdx.x == 0) {
        unsigned int total = 0;
        for(int i = 0; i < gridDim.x; i++) {
            printf("conflict of block %d: %d\n", i, conflict_array.counter[i]);
            total += conflict_array.counter[i];
        }
        printf("total conflict: %d\n", total);
    }
    */
}

inline void set_coo_element(Matrix mat, int index, int i, int j, float val) {
    mat.I[index] = i;
    mat.J[index] = j;
    mat.val[index] = val;
}
void hashtable_to_coo(int COLS, unsigned int *key1, float *val1, thrust::host_vector<unsigned int> key2, thrust::host_vector<float> val2) {
    int nnz = 0;
    for(int i = 0; i < HASH_TABLE_SIZE; i++)
        if(key1[i] != INVALID_KEY) nnz++;
    printf("nnz of C: %d\n", nnz);
    for(int i = 0; i<key2.size(); i++)
        if(key2[i] != INVALID_KEY) nnz++;
    printf("nnz of C: %d\n", nnz);

    Matrix C;
    C.nz = nnz;
    C.I = new int[nnz];
    C.J = new int[nnz];
    C.val = new float[nnz];
    int t = 0;
    for(int i = 0; i < HASH_TABLE_SIZE; i++) {
        unsigned int index = key1[i];
        if(index != INVALID_KEY)
            set_coo_element(C, t++, get_i(index, COLS), get_j(index, COLS), val1[i]);
    }
    for(int i = 0; i < key2.size(); i++) {
        unsigned int index = key2[i];
        if(index != INVALID_KEY)
            set_coo_element(C, t++, get_i(index, COLS), get_j(index, COLS), val2[i]);
    }
}

void esc(Matrix A, Matrix B, int conflict_size) {
    BLOCK_CONFLICT_SIZE = conflict_size;
    printf("\n====================================================\n");
    printf("blocks: %d, threads: %d\nhashtable size: %d, conflict array size: %d\n", GRID_SIZE, BLOCK_SIZE, HASH_TABLE_SIZE, BLOCK_CONFLICT_SIZE);
    printf("====================================================\n\n");
    esc(A, B);
}

// Matrix A: csc format; Matrix B: csr format
void esc(Matrix A, Matrix B) {
    // Allocate device memory and copy matrices from the host to the device
    timer t1;
    Matrix dev_A, dev_B;
    dev_A.M = A.M; dev_A.N = A.N; dev_A.nz = A.nz;
    hipMalloc((void**)&dev_A.I, sizeof(int)*(A.nz));
    hipMalloc((void**)&dev_A.J, sizeof(int)*(A.N+1));
    hipMalloc((void**)&dev_A.val, sizeof(float)*(A.nz));
    hipMemcpy(dev_A.I, A.I, A.nz*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_A.J, A.J, (A.N+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_A.val, A.val, A.nz*sizeof(float), hipMemcpyHostToDevice);
    dev_B.M = B.M; dev_B.N = B.N; dev_B.nz = B.nz;
    hipMalloc((void**)&dev_B.I, sizeof(int)*(B.M+1));
    hipMalloc((void**)&dev_B.J, sizeof(int)*(B.nz));
    hipMalloc((void**)&dev_B.val, sizeof(float)*(B.nz));
    hipMemcpy(dev_B.I, B.I, (B.M+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_B.J, B.J, B.nz*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_B.val, B.val, B.nz*sizeof(float), hipMemcpyHostToDevice);

    // Allocate device memory for has table
    unsigned int *dev_C_key;
    float *dev_C_val;
    hipMalloc((void**)&dev_C_key, sizeof(unsigned int)*HASH_TABLE_SIZE);
    hipMalloc((void**)&dev_C_val, sizeof(float)*HASH_TABLE_SIZE);
    hipMemset(dev_C_val, 0, sizeof(float)*HASH_TABLE_SIZE);

    // Allocate device memory for collision results
    ConflictArray conflict_array;
    hipMalloc((void**)&conflict_array.counter, sizeof(int)*GRID_SIZE);
    hipMalloc((void**)&conflict_array.key, sizeof(unsigned int)*GRID_SIZE * BLOCK_CONFLICT_SIZE);  // each block has 1M for collision results
    hipMalloc((void**)&conflict_array.val, sizeof(float)*GRID_SIZE * BLOCK_CONFLICT_SIZE);
    printf("time for allocating memory: %f milliseconds\n", t1.milliseconds_elapsed());

    timer t2;
    cuda_multiplication<<<GRID_SIZE, BLOCK_SIZE>>>(dev_A, dev_B, dev_C_key, dev_C_val, conflict_array, BLOCK_CONFLICT_SIZE);
    printf("time for SpMM: %f milliseconds\n", t2.milliseconds_elapsed());

    // Copy back the hash table
    unsigned int *C_key = new unsigned int[HASH_TABLE_SIZE];
    float *C_val = new float[HASH_TABLE_SIZE];
    hipMemcpy(C_key, dev_C_key, HASH_TABLE_SIZE*sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(C_val, dev_C_val, HASH_TABLE_SIZE*sizeof(float), hipMemcpyDeviceToHost);

    timer t3;
    thrust::device_ptr<unsigned int> key_ptr(conflict_array.key);
    thrust::device_ptr<float> val_ptr(conflict_array.val);
    thrust::device_vector<unsigned int> output_keys(GRID_SIZE*BLOCK_CONFLICT_SIZE);
    thrust::device_vector<float> output_vals(GRID_SIZE*BLOCK_CONFLICT_SIZE);
    thrust::sort_by_key(key_ptr, key_ptr+GRID_SIZE*BLOCK_CONFLICT_SIZE, val_ptr);
    thrust::reduce_by_key(key_ptr, key_ptr+GRID_SIZE*BLOCK_CONFLICT_SIZE, val_ptr, output_keys.begin(), output_vals.begin(), thrust::equal_to<unsigned int>(), thrust::plus<float>());
    printf("time for reduction: %f milliseconds\n", t3.milliseconds_elapsed());

    thrust::host_vector<unsigned int> key2 = output_keys;
    thrust::host_vector<float> val2 = output_vals;
    hashtable_to_coo(A.N, C_key, C_val, key2, val2);

    hipFree(dev_A.I);
    hipFree(dev_A.J);
    hipFree(dev_A.val);
    hipFree(dev_B.I);
    hipFree(dev_B.J);
    hipFree(dev_B.val);
    hipFree(dev_C_key);
    hipFree(dev_C_val);
    hipFree(conflict_array.counter);
    hipFree(conflict_array.key);
    hipFree(conflict_array.val);
    delete C_key;
    delete C_val;
}
